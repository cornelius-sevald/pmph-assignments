
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

void serial_map(float *in, float *out, unsigned int N) {
    for (unsigned int i = 0; i < N; ++i) {
        float x = in[i];
        float y = (x/(x-2.3))*(x/(x-2.3))*(x/(x-2.3)); // (x/(x-2.3))^3
        out[i] = y;
    }
}

int main(int argc, char** argv) {
    // size of array
    // can be set by command line args but defaults to 753411
    unsigned int N = 753411;
    if (argc > 1) {
        N = strtoul(argv[1], NULL, 10);
        assert(N != 0);
    }
    size_t mem_size = N*sizeof(float);

    // allocate host memory
    float *h_in  = (float *) malloc(mem_size);
    float *h_out = (float *) malloc(mem_size);

    // initialize memory
    for (unsigned int i = 0; i < N; ++i) {
        h_in[i] = (float) (i+1);
    }

    // preform serial map
    serial_map(h_in, h_out, N);

    // print results for debugging
    for (unsigned int i = 0; i < N; ++i) {
        printf("%d\t%.2f\n", i, h_out[i]);
    }
}
