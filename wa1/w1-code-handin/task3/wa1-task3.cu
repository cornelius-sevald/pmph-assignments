
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

void serial_map(float *in, float *out, unsigned int N) {
    for (unsigned int i = 0; i < N; ++i) {
        float x = in[i];
        float y = (x/(x-2.3))*(x/(x-2.3))*(x/(x-2.3)); // (x/(x-2.3))^3
        out[i] = y;
    }
}

__global__ void parallel_map(float *d_in, float *d_out, unsigned int N) {
    const unsigned int lid = threadIdx.x;
    const unsigned int gid = blockIdx.x*blockDim.x + lid;
    if (gid < N) {
        float x = d_in[gid];
        float y = (x/(x-2.3))*(x/(x-2.3))*(x/(x-2.3)); // (x/(x-2.3))^3
        d_out[gid] = y;
    }
}

int main(int argc, char** argv) {
    // size of array
    // can be set by command line args but defaults to 753411
    unsigned int N = 753411;
    if (argc > 1) {
        N = strtoul(argv[1], NULL, 10);
        assert(N != 0);
    }
    size_t mem_size = N*sizeof(float);
    unsigned int block_size = 256;
    unsigned int num_blocks = ((N + (block_size - 1)) / block_size);

    // allocate host memory
    float *h_in  = (float *) malloc(mem_size);
    float *h_out = (float *) malloc(mem_size);

    // initialize memory
    for (unsigned int i = 0; i < N; ++i) {
        h_in[i] = (float) (i+1);
    }

    // allocate device memory
    float *d_in;
    float *d_out;
    hipMalloc((void **) &d_in,  mem_size);
    hipMalloc((void **) &d_out, mem_size);

    // copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    // preform parallel map
    parallel_map<<<num_blocks, block_size>>>(d_in, d_out, N);

    // copy host memory to device
    hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

    // print results for debugging
    for (unsigned int i = 0; i < N; ++i) {
        printf("%d\t%.2f\n", i, h_out[i]);
    }

    // clean up
    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);
}
