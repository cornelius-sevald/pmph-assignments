
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

int main(int argc, char** argv) {
    // Size of array
    // Can be set by command line args but defaults to 753411
    unsigned long N = 753411;

    if (argc > 1) {
        N = strtoul(argv[1], NULL, 10);
        assert(N != 0);
    }

    printf("N = %lu\n", N);
}
