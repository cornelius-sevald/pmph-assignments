#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <assert.h>
#include <sys/time.h>
#include <time.h>

void serial_map(float *in, float *out, unsigned int N) {
    for (unsigned int i = 0; i < N; ++i) {
        float x = in[i];
        float y = (x/(x-2.3))*(x/(x-2.3))*(x/(x-2.3)); // (x/(x-2.3))^3
        out[i] = y;
    }
}

__global__ void parallel_map(float *d_in, float *d_out, unsigned int N) {
    const unsigned int lid = threadIdx.x;
    const unsigned int gid = blockIdx.x*blockDim.x + lid;
    if (gid < N) {
        float x = d_in[gid];
        float y = (x/(x-2.3))*(x/(x-2.3))*(x/(x-2.3)); // (x/(x-2.3))^3
        d_out[gid] = y;
    }
}

bool check_equal(float *arr1, float *arr2, unsigned int N) {
    const float epsilon = 0.00001;
    bool are_equal = true;

    for (unsigned int i = 0; i < N; ++i) {
        are_equal = are_equal && fabs(arr1[i] - arr2[i]) < epsilon;
    }

    return are_equal;
}

int timeval_subtract(
        struct timeval *result,
        struct timeval *t2,
        struct timeval *t1
        ) {
    unsigned int resolution = 1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) -
                    (t1->tv_usec + resolution * t1->tv_sec);
    result->tv_sec  = diff / resolution;
    result->tv_usec = diff % resolution;
    return diff < 0;
}

#define CPU_RUNS 100
#define GPU_RUNS 100

int main(int argc, char** argv) {
    unsigned long int cpu_elapsed, gpu_elapsed;
    struct timeval t_start, t_end, t_diff;

    // size of array
    // can be set by command line args but defaults to 753411
    unsigned int N = 753411;
    if (argc > 1) {
        N = strtoul(argv[1], NULL, 10);
        assert(N != 0);
    }
    size_t mem_size = N*sizeof(float);
    unsigned int block_size = 1024;
    unsigned int num_blocks = ((N + (block_size - 1)) / block_size);

    // allocate host memory
    float *h_in    = (float *) malloc(mem_size);
    float *h_out_s = (float *) malloc(mem_size); // serial output
    float *h_out_p = (float *) malloc(mem_size); // parallel output

    // initialize memory
    for (unsigned int i = 0; i < N; ++i) {
        h_in[i] = (float) (i+1);
    }

    // preform serial map and time it
    gettimeofday(&t_start, NULL);
    for (int i = 0; i < CPU_RUNS; i++) {
        serial_map(h_in, h_out_s, N);
    }
    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    cpu_elapsed = (t_diff.tv_sec*1e6 + t_diff.tv_usec) / CPU_RUNS;

    // allocate device memory
    float *d_in;
    float *d_out;
    hipMalloc((void **) &d_in,  mem_size);
    hipMalloc((void **) &d_out, mem_size);

    // copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    // preform parallel map
    gettimeofday(&t_start, NULL);
    for (int i = 0; i < GPU_RUNS; i++) {
        parallel_map<<<num_blocks, block_size>>>(d_in, d_out, N);
    } hipDeviceSynchronize();
    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    gpu_elapsed = (t_diff.tv_sec*1e6 + t_diff.tv_usec) / GPU_RUNS;

    // copy host memory to device
    hipMemcpy(h_out_p, d_out, mem_size, hipMemcpyDeviceToHost);

    if (check_equal(h_out_s, h_out_p, N)) {
        printf("VALID\n");
    } else {
        printf("INVALID\n");
    }

    printf("CPU runtime %d (ms)\n", cpu_elapsed);
    printf("GPU runtime %d (ms)\n", gpu_elapsed);

    // clean up
    free(h_in);
    free(h_out_s);
    free(h_out_p);
    hipFree(d_in);
    hipFree(d_out);
}
