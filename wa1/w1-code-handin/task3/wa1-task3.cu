
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <assert.h>

void serial_map(float *in, float *out, unsigned int N) {
    for (unsigned int i = 0; i < N; ++i) {
        float x = in[i];
        float y = (x/(x-2.3))*(x/(x-2.3))*(x/(x-2.3)); // (x/(x-2.3))^3
        out[i] = y;
    }
}

__global__ void parallel_map(float *d_in, float *d_out, unsigned int N) {
    const unsigned int lid = threadIdx.x;
    const unsigned int gid = blockIdx.x*blockDim.x + lid;
    if (gid < N) {
        float x = d_in[gid];
        float y = (x/(x-2.3))*(x/(x-2.3))*(x/(x-2.3)); // (x/(x-2.3))^3
        d_out[gid] = y;
    }
}

bool check_equal(float *arr1, float *arr2, unsigned int N) {
    const float epsilon = 0.00001;
    bool are_equal = true;

    for (unsigned int i = 0; i < N; ++i) {
        are_equal = are_equal && fabs(arr1[i] - arr2[i]) < epsilon;
    }

    return are_equal;
}

int main(int argc, char** argv) {
    // size of array
    // can be set by command line args but defaults to 753411
    unsigned int N = 753411;
    if (argc > 1) {
        N = strtoul(argv[1], NULL, 10);
        assert(N != 0);
    }
    size_t mem_size = N*sizeof(float);
    unsigned int block_size = 256;
    unsigned int num_blocks = ((N + (block_size - 1)) / block_size);

    // allocate host memory
    float *h_in    = (float *) malloc(mem_size);
    float *h_out_s = (float *) malloc(mem_size); // serial output
    float *h_out_p = (float *) malloc(mem_size); // parallel output

    // initialize memory
    for (unsigned int i = 0; i < N; ++i) {
        h_in[i] = (float) (i+1);
    }

    // allocate device memory
    float *d_in;
    float *d_out;
    hipMalloc((void **) &d_in,  mem_size);
    hipMalloc((void **) &d_out, mem_size);

    // preform serial map
    serial_map(h_in, h_out_s, N);

    // copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    // preform parallel map
    parallel_map<<<num_blocks, block_size>>>(d_in, d_out, N);

    // copy host memory to device
    hipMemcpy(h_out_p, d_out, mem_size, hipMemcpyDeviceToHost);

    if (check_equal(h_out_s, h_out_p, N)) {
        printf("VALID\n");
    } else {
        printf("INVALID\n");
    }

    // clean up
    free(h_in);
    free(h_out_s);
    free(h_out_p);
    hipFree(d_in);
    hipFree(d_out);
}
